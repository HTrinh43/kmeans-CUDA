#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cfloat>
#include <iostream>
#include "helpers.h"
#include <cmath>
#include <chrono>

__global__ void assignLabels(float *points, float *centroids, int *labels, int n_points, int k, int n_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_points) return;

    int nearestCentroid = 0;
    float minDist = FLT_MAX;

    for (int centroidIdx = 0; centroidIdx < k; centroidIdx++) {
        float dist = 0.0f;

        for (int dim = 0; dim < n_dim; dim++) {
            float diff = points[idx * n_dim + dim] - centroids[centroidIdx * n_dim + dim];
            dist += diff * diff;
        }

        if (dist < minDist) {
            minDist = dist;
            nearestCentroid = centroidIdx;
        }
    }

    labels[idx] = nearestCentroid;
}

void findClosestCentroid(float* points, int* d_labels, int n_points, int n_dim, int k,float* d_centroids) {
    int blockSize = 256;
    int gridSize = (n_points + blockSize - 1) / blockSize;

    // Launching the kernel
    assignLabels<<<gridSize, blockSize>>>(points, d_centroids, d_labels, n_points, k, n_dim);

    // Synchronize to ensure the kernel has completed
    hipDeviceSynchronize();

}


__global__ void checkConvergenceKernel(float* d_centroids, float* d_old_centroids, float* d_diffs, int k, int n_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k * n_dim) {
        float diff = d_centroids[idx] - d_old_centroids[idx];
        d_diffs[idx] = diff * diff;  // store squared difference
    }
}

bool hasConverged(float* d_centroids, float* d_old_centroids, float* d_diffs, int k, int n_dim, float threshold) {
    // Compute squared differences using the kernel
    checkConvergenceKernel<<<(k * n_dim + 255) / 256, 256>>>(d_centroids, d_old_centroids, d_diffs, k, n_dim);
    hipDeviceSynchronize(); // Ensure kernel completion

    // Copy the squared differences to host
    float* h_diffs = new float[k * n_dim];
    hipMemcpy(h_diffs, d_diffs, k * n_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Compute total difference on the host
    float totalDiff = 0.0f;
    for (int i = 0; i < k * n_dim; i++) {
        totalDiff += h_diffs[i];
    }

    delete[] h_diffs; // Clean up

    // Check for convergence and return the result

    return sqrt(totalDiff) < threshold;
}



__global__ void updateCentroidsKernel(float* points, int* labels, float* d_centroids_sums, int* d_counts, int n_points, int n_dim, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_points) {
        for (int d = 0; d < n_dim; d++) {
            atomicAdd(&d_centroids_sums[labels[idx] * n_dim + d], points[idx * n_dim + d]);
        }
        atomicAdd(&d_counts[labels[idx]], 1);
    }
}

__global__ void computeNewCentroidsKernel(float* d_centroids_sums, int* d_counts, float* d_centroids, int n_dim, int k) {
    int centroid_idx = blockIdx.x;
    int dim = threadIdx.x;

    if (d_counts[centroid_idx] > 0) {  // Avoid division by zero
        d_centroids[centroid_idx * n_dim + dim] = d_centroids_sums[centroid_idx * n_dim + dim] / d_counts[centroid_idx];
    }
    else {
        d_centroids[centroid_idx * n_dim + dim] = 0;
    }
}

void updateCentroids(float* d_points,int* d_labels, float* d_centroids, int n_points, int n_dim, int k, 
float* d_centroids_sums, int* d_counts) {
    
    hipMemset(d_centroids_sums, 0, k * n_dim * sizeof(float));
    hipMemset(d_counts, 0, k * sizeof(int));
    int blockSize = 256;  // Or whatever number is optimal for your GPU and problem.
    int gridSize = (n_points + blockSize - 1) / blockSize;


    // Sum up the points for each centroid
    updateCentroidsKernel<<<gridSize, blockSize>>>(d_points, d_labels, d_centroids_sums, d_counts, n_points, n_dim, k);
    hipDeviceSynchronize();

    // Compute new centroids
    computeNewCentroidsKernel<<<k, n_dim>>>(d_centroids_sums, d_counts, d_centroids, n_dim, k);
    hipDeviceSynchronize();

    // hipFree(d_centroids_sums);
    // hipFree(d_counts);
}


float* kmeansCUDA(Point_cu data, kmeans_args_t args) {


    // Create CUDA events for timing
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record the start event
    hipEventRecord(start, 0);

    // Dynamic allocation of device-side variables
    int k = args.k;
    int n_dim = args.d;
    int n_val = data.size;
    int max_iters = args.m;
    float threshold = args.t;
    int seed = args.s;

    float* d_points;
    float* d_centroids;
    float* d_old_centroids;
    int* d_labels;
    hipMalloc(&d_points, n_val * n_dim * sizeof(float));
    hipMalloc(&d_centroids, k * n_dim * sizeof(float));
    hipMalloc(&d_old_centroids, k * n_dim * sizeof(float));
    hipMalloc(&d_labels, n_val * sizeof(int));

    float* d_centroids_sums;
    int* d_counts;

    hipMalloc(&d_centroids_sums, k * n_dim * sizeof(float));
    hipMalloc(&d_counts, k * sizeof(int));
    // Initialize centroids on host
    float* h_centroids = initializeCentroids(data, n_val,n_dim, k, seed);
    float* d_diffs;

    auto start1 = std::chrono::high_resolution_clock::now();
    hipMalloc((void**)&d_diffs, k * n_dim * sizeof(float));

    // Transfer centroids to device
    hipMemcpy(d_centroids, h_centroids, k * n_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_points, data.features, n_val * n_dim * sizeof(float), hipMemcpyHostToDevice);
    // const int threadsPerBlock = 128;
    // const int blocks = (n_val + threadsPerBlock - 1) / threadsPerBlock;
    hipMemcpy(d_old_centroids, d_centroids, k * n_dim * sizeof(float), hipMemcpyDeviceToDevice);
    int* h_labels = new int[n_val];
    int iter = 0;

    auto end1 = std::chrono::high_resolution_clock::now();
    double avgTime1 = std::chrono::duration<double, std::milli>(end1 - start1).count();
    printf("%lf\n", avgTime1);
    while (iter < max_iters) {

        
        findClosestCentroid(d_points, d_labels, n_val, n_dim, k, d_centroids);

        updateCentroids(d_points,d_labels, d_centroids, n_val, n_dim, k, d_centroids_sums, d_counts);
        iter++;
        hipMemset(d_diffs, 0, k * n_dim * sizeof(float));

        if (hasConverged(d_centroids, d_old_centroids, d_diffs, k, n_dim, threshold)) {
            break;  // Exit loop if centroids have converged
        }

        // Swap the pointers
        hipMemcpy(d_old_centroids, d_centroids, k * n_dim * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipMemcpy(h_centroids, d_centroids, k * n_dim * sizeof(float), hipMemcpyDeviceToHost);
    
    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    // Compute the elapsed time between start and stop
    hipEventElapsedTime(&elapsedTime, start, stop);
    double avgTime = elapsedTime / iter;
    printf("%d,%lf\n", iter, avgTime);
    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Cleanup
    hipFree(d_centroids);
    hipFree(d_old_centroids);
    hipFree(data.label);
    hipFree(d_diffs);


    return h_centroids;
}
